#include "hip/hip_runtime.h"
#include "kernel.h"

#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>

#include <hip/hip_runtime.h>
#include <stdlib.h>

#include <iostream>

int main(int argc, char** argv)
{
    std::string str(argv[1]);
    std::cout << str << std::endl;
    cv::Mat img = cv::imread(str, cv::IMREAD_COLOR);
    if(img.empty())
    {
        std::cout << "Could not read the image: " << str << std::endl;
        return 1;
    }

    cv::namedWindow("Original", 0);
    cv::resizeWindow("Original", 1280,720);
    cv::imshow("Original", img);
    
    const int w = img.cols;
    const int h = img.rows;

    if(*argv[2] == '0')
    {
        uchar4 *arrS = (uchar4*)malloc(w*h*sizeof(uchar4));
        cv::Mat sharp = img.clone();
        uint8_t* pixelPtr = (uint8_t*)img.data;
        int cn = img.channels();
        for (int r = 0; r < h; ++r) {
            for (int c = 0; c < w; ++c) {
                arrS[r*w + c].x = pixelPtr[r*img.cols*cn + c*cn + 0];
                arrS[r*w + c].y = pixelPtr[r*img.cols*cn + c*cn + 1];
                arrS[r*w + c].z = pixelPtr[r*img.cols*cn + c*cn + 2];
            }
        }

        sharpenParallel(arrS, w, h);

        for (int r = 0; r < h; ++r) {
            for (int c = 0; c < w; ++c) {
                sharp.at<uchar>(r, c, 0) = arrS[r*w + c].x;
                sharp.at<uchar>(r, c, 1) = arrS[r*w + c].y;
                sharp.at<uchar>(r, c, 2) = arrS[r*w + c].z;
            }
        }
        
        
        cv::namedWindow("Sharpen", 0);
        cv::resizeWindow("Sharpen", 1280,720);
        cv::imshow("Sharpen", sharp);
        cv::waitKey(0);
        free(arrS);
    }
    else if(*argv[2] == '1')
    {
        uchar4 *arrI = (uchar4*)malloc(w*h*sizeof(uchar4));

        uint8_t* pixelPtr = (uint8_t*)img.data;
        int cn = img.channels();
        for (int r = 0; r < h; ++r) {
            for (int c = 0; c < w; ++c) {
                arrI[r*w + c].x = pixelPtr[r*img.cols*cn + c*cn + 0];
                arrI[r*w + c].y = pixelPtr[r*img.cols*cn + c*cn + 1];
                arrI[r*w + c].z = pixelPtr[r*img.cols*cn + c*cn + 2];
            }
        }

        unsigned char* outArr = (unsigned char*)malloc(w*h*sizeof(unsigned char));
        intensityParallel(arrI, outArr, w, h);
        cv::Mat intens = cv::Mat(img.rows,img.cols, CV_8UC1, (void*)outArr);

        cv::namedWindow("Intensity", 0);
        cv::resizeWindow("Intensity", 1280,720);
        cv::imshow("Intensity", intens);
        cv::waitKey(0);
        free(arrI);
        free(outArr);
    }
    else if(false)
    {
        unsigned char* outArr = (unsigned char*)malloc(w*h*sizeof(unsigned char));
        uint8_t* pixelPtr = (uint8_t*)img.data;
        int cn = img.channels();
        for (int r = 0; r < h; ++r) {
            for (int c = 0; c < w; ++c) {
                outArr[r*w + c] = 0.2126*pixelPtr[r*img.cols*cn + c*cn + 0] + 0.7152*pixelPtr[r*img.cols*cn + c*cn + 1]+0.0722*pixelPtr[r*img.cols*cn + c*cn + 2];
            }
        }

        horizSobelParallel(outArr, w, h);

        cv::Mat sobel = cv::Mat(img.rows,img.cols, CV_8UC1, (void*)outArr);
        
        cv::namedWindow("Sobel", 0);
        cv::resizeWindow("Sobel", 1280,720);
        cv::imshow("Sobel", sobel);
        cv::waitKey(0);
        free(outArr);
    }

    return 0;
}