#include "hip/hip_runtime.h"
#include "kernel.h"
#define TX 32
#define TY 32
#define LEN 10.f
#define TIME_STEP 0.005f
#define FINAL_TIME 10.f
#define M 2.0f
#define G 9.81f
#define R 2.f

__device__
float scale(int i, int w) { return 2 * LEN*(((1.f*i)/w) - 0.5f); }

__device__
float f(float x, float y, float dampening, float sys) {
  if (sys == 1) return x - 2 * dampening * y; 
  if (sys == 2) return (-dampening*y/M*R*R)-(G/R*sin(x)); 
  else return -x - 2 * dampening * y;
}

__device__
float2 euler(float x, float y, float dt, float tFinal,
             float dampening, float sys) {
  float dx = 0.f, dy = 0.f;
  for (float t = 0; t < tFinal; t += dt) {
    dx = dt*y;
    dy = dt*f(x, y, dampening, sys);
    x += dx;
    y += dy;
  }
  return make_float2(x, y);
}

__device__
unsigned char clip(float x){ return x > 255 ? 255 : (x < 0 ? 0 : x); }

__global__
void phaseSpaceImageKernel(uchar4 *d_out, int w, int h, float d, int s) {
  const int c = blockIdx.x*blockDim.x + threadIdx.x;
  const int r = blockIdx.y*blockDim.y + threadIdx.y;

  if ((c >= w) || (r >= h)) return; 
  
  const int i = c + r*w;
  const float x0 = scale(c, w);
  const float y0 = scale(r, h);
  const float dist_0 = sqrt(x0*x0 + y0*y0);
  const float2 pos = euler(x0, y0, TIME_STEP, FINAL_TIME, d, s);
  const float dist_f = sqrt(pos.x*pos.x + pos.y*pos.y);

  const float dist_r = dist_f / dist_0;
  d_out[i].x = clip(dist_r * 255); // red ~ growth
  d_out[i].y = ((c == w / 2) || (r == h / 2)) ? 255 : 0; // axes
  d_out[i].z = clip((1 / dist_r) * 255); // blue ~ 1/growth
  d_out[i].w = 255;
}

void kernelLauncher(uchar4 *d_out, int w, int h, float d, int s) {
  const dim3 blockSize(TX, TY);
  const dim3 gridSize = dim3((w + TX - 1)/TX, (h + TY - 1)/TY);
  phaseSpaceImageKernel<<<gridSize, blockSize >>>(d_out, w, h, d, s);
}
